#include "hip/hip_runtime.h"
#include "yolo.hpp"

namespace yolo {

using namespace std;

#define GPU_BLOCK_THREADS 512
#define checkRuntime(call)                                                                 \
  do {                                                                                     \
    auto ___call__ret_code__ = (call);                                                     \
    if (___call__ret_code__ != hipSuccess) {                                              \
      INFO("CUDA Runtime error💥 %s # %s, code = %s [ %d ]", #call,                        \
           hipGetErrorString(___call__ret_code__), hipGetErrorName(___call__ret_code__), \
           ___call__ret_code__);                                                           \
      abort();                                                                             \
    }                                                                                      \
  } while (0)

#define checkKernel(...)                 \
  do {                                   \
    {                                    \
      (__VA_ARGS__);                     \
    }                                    \
    checkRuntime(hipPeekAtLastError()); \
  } while (0)

const int NUM_BOX_ELEMENT = 8;  // left, top, right, bottom, confidence, class,
                                // keepflag, row_index(output)
const int MAX_IMAGE_BOXES = 1024;
inline int upbound(int n, int align = 32) { return (n + align - 1) / align * align; }
static __host__ __device__ void affine_project(float *matrix, float x, float y, float *ox,
                                               float *oy) {
  *ox = matrix[0] * x + matrix[1] * y + matrix[2];
  *oy = matrix[3] * x + matrix[4] * y + matrix[5];
}

static __global__ void decode_kernel_common(float *predict, int num_bboxes, int num_classes,
                                            int output_cdim, float confidence_threshold,
                                            float *invert_affine_matrix, float *parray,
                                            int MAX_IMAGE_BOXES) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= num_bboxes) return;

  float *pitem = predict + output_cdim * position;
  float objectness = pitem[4];
  if (objectness < confidence_threshold) return;

  float *class_confidence = pitem + 5;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_classes; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }

  confidence *= objectness;
  if (confidence < confidence_threshold) return;

  int index = atomicAdd(parray, 1);
  if (index >= MAX_IMAGE_BOXES) return;

  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;
  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  affine_project(invert_affine_matrix, left, top, &left, &top);
  affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

  float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;  // 1 = keep, 0 = ignore
}

static __global__ void decode_kernel_v8(float *predict, int num_bboxes, int num_classes,
                                        int output_cdim, float confidence_threshold,
                                        float *invert_affine_matrix, float *parray,
                                        int MAX_IMAGE_BOXES) {
  int position = blockDim.x * blockIdx.x + threadIdx.x;
  if (position >= num_bboxes) return;

  float *pitem = predict + output_cdim * position;
  float *class_confidence = pitem + 4;
  float confidence = *class_confidence++;
  int label = 0;
  for (int i = 1; i < num_classes; ++i, ++class_confidence) {
    if (*class_confidence > confidence) {
      confidence = *class_confidence;
      label = i;
    }
  }
  if (confidence < confidence_threshold) return;

  int index = atomicAdd(parray, 1);
  if (index >= MAX_IMAGE_BOXES) return;

  float cx = *pitem++;
  float cy = *pitem++;
  float width = *pitem++;
  float height = *pitem++;
  float left = cx - width * 0.5f;
  float top = cy - height * 0.5f;
  float right = cx + width * 0.5f;
  float bottom = cy + height * 0.5f;
  affine_project(invert_affine_matrix, left, top, &left, &top);
  affine_project(invert_affine_matrix, right, bottom, &right, &bottom);

  float *pout_item = parray + 1 + index * NUM_BOX_ELEMENT;
  *pout_item++ = left;
  *pout_item++ = top;
  *pout_item++ = right;
  *pout_item++ = bottom;
  *pout_item++ = confidence;
  *pout_item++ = label;
  *pout_item++ = 1;  // 1 = keep, 0 = ignore
  *pout_item++ = position;
}

static __device__ float box_iou(float aleft, float atop, float aright, float abottom, float bleft,
                                float btop, float bright, float bbottom) {
  float cleft = max(aleft, bleft);
  float ctop = max(atop, btop);
  float cright = min(aright, bright);
  float cbottom = min(abottom, bbottom);

  float c_area = max(cright - cleft, 0.0f) * max(cbottom - ctop, 0.0f);
  if (c_area == 0.0f) return 0.0f;

  float a_area = max(0.0f, aright - aleft) * max(0.0f, abottom - atop);
  float b_area = max(0.0f, bright - bleft) * max(0.0f, bbottom - btop);
  return c_area / (a_area + b_area - c_area);
}

static __global__ void fast_nms_kernel(float *bboxes, int MAX_IMAGE_BOXES, float threshold) {
  int position = (blockDim.x * blockIdx.x + threadIdx.x);
  int count = min((int)*bboxes, MAX_IMAGE_BOXES);
  if (position >= count) return;

  // left, top, right, bottom, confidence, class, keepflag
  float *pcurrent = bboxes + 1 + position * NUM_BOX_ELEMENT;
  for (int i = 0; i < count; ++i) {
    float *pitem = bboxes + 1 + i * NUM_BOX_ELEMENT;
    if (i == position || pcurrent[5] != pitem[5]) continue;

    if (pitem[4] >= pcurrent[4]) {
      if (pitem[4] == pcurrent[4] && i < position) continue;

      float iou = box_iou(pcurrent[0], pcurrent[1], pcurrent[2], pcurrent[3], pitem[0], pitem[1],
                          pitem[2], pitem[3]);

      if (iou > threshold) {
        pcurrent[6] = 0;  // 1=keep, 0=ignore
        return;
      }
    }
  }
}

static dim3 grid_dims(int numJobs) {
  int numBlockThreads = numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
  return dim3(((numJobs + numBlockThreads - 1) / (float)numBlockThreads));
}

static dim3 block_dims(int numJobs) {
  return numJobs < GPU_BLOCK_THREADS ? numJobs : GPU_BLOCK_THREADS;
}

static void decode_kernel_invoker(float *predict, int num_bboxes, int num_classes, int output_cdim,
                                  float confidence_threshold, float nms_threshold,
                                  float *invert_affine_matrix, float *parray, int MAX_IMAGE_BOXES,
                                  Type type, hipStream_t stream) {
  auto grid = grid_dims(num_bboxes);
  auto block = block_dims(num_bboxes);

  if (type == Type::V8 || type == Type::V8Seg) {
    checkKernel(decode_kernel_v8<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
  } else {
    checkKernel(decode_kernel_common<<<grid, block, 0, stream>>>(
        predict, num_bboxes, num_classes, output_cdim, confidence_threshold, invert_affine_matrix,
        parray, MAX_IMAGE_BOXES));
  }

  grid = grid_dims(MAX_IMAGE_BOXES);
  block = block_dims(MAX_IMAGE_BOXES);
  checkKernel(fast_nms_kernel<<<grid, block, 0, stream>>>(parray, MAX_IMAGE_BOXES, nms_threshold));
}

static __global__ void decode_single_mask_kernel(int left, int top, float *mask_weights,
                                                 float *mask_predict, int mask_width,
                                                 int mask_height, unsigned char *mask_out,
                                                 int mask_dim, int out_width, int out_height) {
  // mask_predict to mask_out
  // mask_weights @ mask_predict
  int dx = blockDim.x * blockIdx.x + threadIdx.x;
  int dy = blockDim.y * blockIdx.y + threadIdx.y;
  if (dx >= out_width || dy >= out_height) return;

  int sx = left + dx;
  int sy = top + dy;
  if (sx < 0 || sx >= mask_width || sy < 0 || sy >= mask_height) {
    mask_out[dy * out_width + dx] = 0;
    return;
  }

  float cumprod = 0;
  for (int ic = 0; ic < mask_dim; ++ic) {
    float cval = mask_predict[(ic * mask_height + sy) * mask_width + sx];
    float wval = mask_weights[ic];
    cumprod += cval * wval;
  }

  float alpha = 1.0f / (1.0f + exp(-cumprod));
  mask_out[dy * out_width + dx] = alpha * 255;
}

static void decode_single_mask(float left, float top, float *mask_weights, float *mask_predict,
                               int mask_width, int mask_height, unsigned char *mask_out,
                               int mask_dim, int out_width, int out_height, hipStream_t stream) {
  // mask_weights is mask_dim(32 element) gpu pointer
  dim3 grid((out_width + 31) / 32, (out_height + 31) / 32);
  dim3 block(32, 32);

  checkKernel(decode_single_mask_kernel<<<grid, block, 0, stream>>>(
      left, top, mask_weights, mask_predict, mask_width, mask_height, mask_out, mask_dim, out_width,
      out_height));
}

const char *type_name(Type type) {
  switch (type) {
    case Type::V5:
      return "YoloV5";
    case Type::V3:
      return "YoloV3";
    case Type::V7:
      return "YoloV7";
    case Type::X:
      return "YoloX";
    case Type::V8:
      return "YoloV8";
    default:
      return "Unknow";
  }
}

};  // namespace yolo